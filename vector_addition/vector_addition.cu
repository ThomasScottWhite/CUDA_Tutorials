#include <hip/hip_runtime.h>
#include <stdio.h>

unsigned int cdiv(int a, int b)
{
    return (a + b - 1) / b;
}

__global__ void vecAddKernal(float *A, float *B, float *C, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}
void vecAdd(float *A, float *B, float *C, int n)
{
    float *A_d, *B_d, *C_d;

    size_t size = n * sizeof(float);

    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

    const unsigned int numThreads = 256;
    unsigned int numBlocks = cdiv(n, numThreads);

    vecAddKernal<<<numBlocks, numThreads>>>(A_d, B_d, C_d, n);

    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}
int main()
{
    const int n = 1000;

    float A[n];
    float B[n];
    float C[n];

    for (int i = 0; i < n; i++)
    {
        A[i] = float(i);
        B[i] = float(i);
    }

    vecAdd(A, B, C, n);

    for (int i = 0; i < n; i += 1)
    {
        if (i > 0)
        {
            printf(", ");
        }
        if (i % 10 == 0)
        {
            printf("\n");
        }
        printf("%8.3f", C[i]);
    }
    printf("\n");
    return 0;
}